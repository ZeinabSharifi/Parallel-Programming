#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define TILE 1024
//#define TILE 4

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

__global__ void kernelReverseBit2(float* x_r_d, float* x_i_d, const unsigned int M){
	int i = (TILE/2)*bx + tx;
	float tmp_r,tmp_i;
	unsigned int count ;
	unsigned int index, tmp_index ;
	unsigned int reverse_index;
	for(int j=0; j<8; j++){
		index = 8*i+j;
		tmp_index = index;
		count = M;
		reverse_index = 0;
		while(tmp_index)
		{
			reverse_index <<= 1; 
			reverse_index |= tmp_index & 1;
			tmp_index >>= 1;
			count--;
		}
		reverse_index <<= count;

		//printf("index is: %d and reverse_index is %d\n", 8*i+j, reverse_index);

		if(reverse_index > index){
			tmp_r = x_r_d[index];
			tmp_i = x_i_d[index];
			x_r_d[index] = x_r_d[reverse_index];
			x_i_d[index] = x_i_d[reverse_index];
			x_r_d[reverse_index] = tmp_r;
			x_i_d[reverse_index] = tmp_i;
		}
	}

}

__global__ void kernelReverseBit4(float* x_r_d, float* x_i_d, const unsigned int M){
	int i = (TILE/2)*bx + tx;
	float tmp_r,tmp_i;
	unsigned int count ;
	unsigned int index, tmp_index ;
	unsigned int reverse_index;
	for(int j=0; j<8; j++){
		index = 8*i+j;
		tmp_index = index;
		count = M/2;
		reverse_index = 0;
		while(tmp_index)
		{
			reverse_index <<= 2; 
			reverse_index |= tmp_index & 3;
			tmp_index >>= 2;
			count--;
		}
		reverse_index <<= 2*count;

		//printf("index is: %d and reverse_index is %d\n", 8*i+j, reverse_index);

		if(reverse_index > index){
			tmp_r = x_r_d[index];
			tmp_i = x_i_d[index];
			x_r_d[index] = x_r_d[reverse_index];
			x_i_d[index] = x_i_d[reverse_index];
			x_r_d[reverse_index] = tmp_r;
			x_i_d[reverse_index] = tmp_i;
		}
	}
}

__global__ void kernelFFT2Shared(float* x_r_d, float* x_i_d){
	__shared__ float x_r_s[TILE];
	__shared__ float x_i_s[TILE];

	int i = TILE*bx + tx;

	int index;
	float angle;
	float w_r,w_i,tmp_r1,tmp_r2,tmp_i1,tmp_i2;

	x_r_s[tx] = x_r_d[i];
	x_r_s[tx+TILE/2] = x_r_d[i+TILE/2];
	x_i_s[tx] = x_i_d[i];
	x_i_s[tx+TILE/2] = x_i_d[i+TILE/2];



	for(int j=1; j<TILE; j*=2){
		__syncthreads();
		/*
		if(j==1){
			printf("x_r_s[%d] %f x_r_s[%d] %f\n",tx,x_r_s[tx],tx+TILE/2,x_r_s[tx+TILE/2]);
		}
		
		*/
		index = 2*j*(tx/j) + (tx%j) ;
		angle = index%(2*j)*((float)1/(2*j));
		w_r = cos(2*PI*angle);
		w_i = -sin(2*PI*angle);
		//printf("w_r %f w_i %f\n",w_r,w_i);

		//printf("step %d thread %d index %d angle %f\n",j,i,index,angle );

		tmp_r1 = x_r_s[index];
		tmp_i1 = x_i_s[index];
		tmp_r2 = x_r_s[index+j]*w_r -  x_i_s[index+j]*w_i;
		tmp_i2 = x_r_s[index+j]*w_i +  x_i_s[index+j]*w_r;
		//printf("tmp_r1 %f tmp_r2 %f\n",tmp_r1,tmp_r2);

		x_r_s[index] = tmp_r1 + tmp_r2;
		x_i_s[index] = tmp_i1 + tmp_i2;
		x_r_s[index+j] = tmp_r1 - tmp_r2;
		x_i_s[index+j] = tmp_i1 - tmp_i2;
		//printf("x_r_s[%d] %f x_r_s[%d] %f\n",index,x_r_s[index],index+j,x_r_s[index+j]);

	}
	__syncthreads();

	x_r_d[i] = x_r_s[tx];
	x_i_d[i] = x_i_s[tx];
	x_r_d[i+TILE/2] = x_r_s[tx+TILE/2];
	x_i_d[i+TILE/2] = x_i_s[tx+TILE/2];
}

__global__ void kernelFFT2Global(float* x_r_d, float* x_i_d, const unsigned int j){


	int i = bx*(TILE/2)+ tx;

	int index;
	float angle;
	float w_r, w_i,tmp_r1,tmp_r2,tmp_i1,tmp_i2;


	index = (i%j) + (i/j)*j*2 ;
	//printf("thread %d index %d\n",i,index);
	angle = index%(2*j)*((float)1/(2*j));
	w_r = cos(2*PI*angle);
	w_i = -sin(2*PI*angle);
	//printf("thread %d index %d\n");

	tmp_r1 = x_r_d[index];
	tmp_i1 = x_i_d[index];
	tmp_r2 = x_r_d[index+j]*w_r -  x_i_d[index+j]*w_i;
	tmp_i2 = x_r_d[index+j]*w_i +  x_i_d[index+j]*w_r;

	x_r_d[index] = tmp_r1 + tmp_r2;
	x_i_d[index] = tmp_i1 + tmp_i2;
	x_r_d[index+j] = tmp_r1 - tmp_r2;
	x_i_d[index+j] = tmp_i1 - tmp_i2;

}

__global__ void kernelFFT4Shared(float* x_r_d, float* x_i_d){
	__shared__ float x_r_s[TILE];
	__shared__ float x_i_s[TILE];

	int i = by*gridDim.x*TILE + bx*TILE + tx;
	//int i = bx*TILE + tx;


	int index;
	float angle;
	float w_r1,w_i1, w_r2,w_i2, w_r3,w_i3;
	float tmp_r1,tmp_i1,tmp_r2,tmp_i2, tmp_r3,tmp_i3,tmp_r4,tmp_i4;

	x_r_s[tx] = x_r_d[i];
	x_i_s[tx] = x_i_d[i];
	x_r_s[tx+TILE/4] = x_r_d[i+TILE/4];
	x_i_s[tx+TILE/4] = x_i_d[i+TILE/4];
	x_r_s[tx+TILE/2] = x_r_d[i+TILE/2];
	x_i_s[tx+TILE/2] = x_i_d[i+TILE/2];
	x_r_s[tx+3*TILE/4] = x_r_d[i+3*TILE/4];
	x_i_s[tx+3*TILE/4] = x_i_d[i+3*TILE/4];



	for(int j=1; j<TILE; j*=4){
		__syncthreads();
		/*
		if(j==1){
			printf("x_r_s[%d] %f x_r_s[%d] %f\n",tx,x_r_s[tx],tx+TILE/2,x_r_s[tx+TILE/2]);
		}
		
		*/
		index = 4*j*(tx/j) + (tx%j) ;
		angle = index%(4*j)*((float)1/(4*j));

		w_r1 = cos(2*PI*angle);
		w_i1 = -sin(2*PI*angle);
		w_r2 = cos(4*PI*angle);
		w_i2 = -sin(4*PI*angle);
		w_r3 = cos(6*PI*angle);
		w_i3 = -sin(6*PI*angle);

		//printf("w_r %f w_i %f\n",w_r,w_i);

		//printf("step %d thread %d index %d angle %f\n",j,i,index,angle );

		tmp_r1 = x_r_s[index];
		tmp_i1 = x_i_s[index];
		tmp_r2 = x_r_s[index+j]*w_r1 -  x_i_s[index+j]*w_i1;
		tmp_i2 = x_r_s[index+j]*w_i1 +  x_i_s[index+j]*w_r1;
		tmp_r3 = x_r_s[index+2*j]*w_r2 -  x_i_s[index+2*j]*w_i2;
		tmp_i3 = x_r_s[index+2*j]*w_i2 +  x_i_s[index+2*j]*w_r2;
		tmp_r4 = x_r_s[index+3*j]*w_r3 -  x_i_s[index+3*j]*w_i3;
		tmp_i4 = x_r_s[index+3*j]*w_i3 +  x_i_s[index+3*j]*w_r3;
		//printf("tmp_r1 %f tmp_r2 %f\n",tmp_r1,tmp_r2);

		x_r_s[index] = tmp_r1 + tmp_r2 + tmp_r3 + tmp_r4;
		x_i_s[index] = tmp_i1 + tmp_i2 + tmp_i3 + tmp_i4;

		x_r_s[index+j] = tmp_r1 + tmp_i2 - tmp_r3 - tmp_i4;
		x_i_s[index+j] = tmp_i1 - tmp_r2 - tmp_i3 + tmp_r4;

		x_r_s[index+2*j] = tmp_r1 - tmp_r2 + tmp_r3 - tmp_r4;
		x_i_s[index+2*j] = tmp_i1 - tmp_i2 + tmp_i3 - tmp_i4;

		x_r_s[index+3*j] = tmp_r1 - tmp_i2 - tmp_r3 + tmp_i4;
		x_i_s[index+3*j] = tmp_i1 + tmp_r2 - tmp_i3 - tmp_r4;		
		//printf("x_r_s[%d] %f x_r_s[%d] %f\n",index,x_r_s[index],index+j,x_r_s[index+j]);

	}
	__syncthreads();

	x_r_d[i] = x_r_s[tx];
	x_i_d[i] = x_i_s[tx];
	x_r_d[i+TILE/4] = x_r_s[tx+TILE/4];
	x_i_d[i+TILE/4] = x_i_s[tx+TILE/4];
	x_r_d[i+TILE/2] = x_r_s[tx+TILE/2];
	x_i_d[i+TILE/2] = x_i_s[tx+TILE/2];	
	x_r_d[i+3*TILE/4] = x_r_s[tx+3*TILE/4];
	x_i_d[i+3*TILE/4] = x_i_s[tx+3*TILE/4];
}

__global__ void kernelFFT4Global(float* x_r_d, float* x_i_d, const unsigned int j){


	int i = bx*(TILE/2) + tx;

	int index;
	float angle;
	float w_r1,w_i1, w_r2,w_i2, w_r3,w_i3;
	float tmp_r1,tmp_i1,tmp_r2,tmp_i2, tmp_r3,tmp_i3,tmp_r4,tmp_i4;


	index = 4*j*(i/j) + (i%j) ;
	angle = index%(4*j)*((float)1/(4*j));
	//printf("step %d thread %d index %d angle %f\n",j,i,index,angle );

	w_r1 = cos(2*PI*angle);
	w_i1 = -sin(2*PI*angle);
	w_r2 = cos(4*PI*angle);
	w_i2 = -sin(4*PI*angle);
	w_r3 = cos(6*PI*angle);
	w_i3 = -sin(6*PI*angle);
	//printf("i %d w_r1 %f w_i1 %f\n",i,w_r1,w_i1);
	//printf("i %d w_r2 %f w_i2 %f\n",i,w_r2,w_i2);
	//printf("i %d w_r3 %f w_i3 %f\n",i,w_r3,w_i3);

	

	tmp_r1 = x_r_d[index];
	tmp_i1 = x_i_d[index];
	tmp_r2 = x_r_d[index+j]*w_r1 -  x_i_d[index+j]*w_i1;
	tmp_i2 = x_r_d[index+j]*w_i1 +  x_i_d[index+j]*w_r1;
	tmp_r3 = x_r_d[index+2*j]*w_r2 -  x_i_d[index+2*j]*w_i2;
	tmp_i3 = x_r_d[index+2*j]*w_i2 +  x_i_d[index+2*j]*w_r2;
	tmp_r4 = x_r_d[index+3*j]*w_r3 -  x_i_d[index+3*j]*w_i3;
	tmp_i4 = x_r_d[index+3*j]*w_i3 +  x_i_d[index+3*j]*w_r3;

	//printf("i %d tmp_r1 %f tmp_i1 %f\n",i,tmp_r1,tmp_i1);
	//printf("i %d tmp_r2 %f tmp_i2 %f\n",i,tmp_r2,tmp_i2);
	//printf("i %d tmp_r3 %f tmp_i3 %f\n",i,tmp_r3,tmp_i3);
	//printf("i %d tmp_r4 %f tmp_i4 %f\n",i,tmp_r4,tmp_i4);


	x_r_d[index] = tmp_r1 + tmp_r2 + tmp_r3 + tmp_r4;
	x_i_d[index] = tmp_i1 + tmp_i2 + tmp_i3 + tmp_i4;

	x_r_d[index+j] = tmp_r1 + tmp_i2 - tmp_r3 - tmp_i4;
	x_i_d[index+j] = tmp_i1 - tmp_r2 - tmp_i3 + tmp_r4;

	x_r_d[index+2*j] = tmp_r1 - tmp_r2 + tmp_r3 - tmp_r4;
	x_i_d[index+2*j] = tmp_i1 - tmp_i2 + tmp_i3 - tmp_i4;

	x_r_d[index+3*j] = tmp_r1 - tmp_i2 - tmp_r3 + tmp_i4;
	x_i_d[index+3*j] = tmp_i1 + tmp_r2 - tmp_i3 - tmp_r4;		

	//printf("x_r_d[%d] %f x_i_d[%d] %f\n",index,x_r_d[index],index,x_i_d[index]);
	//printf("x_r_d[%d] %f x_i_d[%d] %f\n",index+j,x_r_d[index+j],index+j,x_i_d[index+j]);
	//printf("x_r_d[%d] %f x_i_d[%d] %f\n",index+2*j,x_r_d[index+2*j],index+2*j,x_i_d[index+2*j]);
	//printf("x_r_d[%d] %f x_i_d[%d] %f\n",index+3*j,x_r_d[index+3*j],index+3*j,x_i_d[index+3*j]);	


}



//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M)
{
	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.

	if(M%2 == 1){ // M = 23 or 25
		kernelReverseBit2<<< N/(4*TILE), TILE/2 >>>(x_r_d, x_i_d, M); // each thread reverses 8 indices
		
		kernelFFT2Shared<<<N/TILE,TILE/2>>>(x_r_d, x_i_d);
		
		for(int j = TILE; j < N; j*=2){
			kernelFFT2Global<<<N/TILE,TILE/2>>>(x_r_d, x_i_d, j);
		}
	}else{ // M = 24 or 26

		kernelReverseBit4<<< N/(4*TILE), TILE/2 >>>(x_r_d, x_i_d, M);// each thread reverses 8 indices

		dim3 dimGrid(N/(2*TILE),2);
		kernelFFT4Shared<<<dimGrid,TILE/4>>>(x_r_d, x_i_d);

		for(int j = TILE; j < N; j*=4){
			kernelFFT4Global<<<N/(2*TILE),TILE/2>>>(x_r_d, x_i_d, j);
		}
		
	}
	

}
